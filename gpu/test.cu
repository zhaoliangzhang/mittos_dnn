
#include <iostream>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

//#define DEVICE_TEST

using namespace std;

#ifdef DEVICE_TEST
int main()
{
    int deviceCount;
    cudaGetDeviceCount(&deviceCount);
    for(int i=0;i<deviceCount;i++)
    {
        cudaDeviceProp devProp;
        cudaGetDeviceProperties(&devProp, i);
        std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
        std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
        std::cout << "======================================================" << std::endl;     
        
    }
    return 0;
}
#else

inline double seconds()
{
    struct timeval tp;
    struct timezone tzp;
    int i = gettimeofday(&tp, &tzp);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

__global__ void CUDA_matrix_multiplication(float Input[], float Weight[], float Output[], int shape[]) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int colum = blockIdx.x*blockDim.x + threadIdx.x;
    
    float sum = 0;
    if(row<shape[0] && colum<shape[2]) {
        for (int i=0; i<shape[1]; i++) {
            sum += Input[row*shape[1] + i] * Weight[i*shape[2] + colum];
        }
        Output[row*shape[2] + colum] = sum;
    }
}

int main() {
    float *input, *weight, *output;
    float *Input, *Weight, *Output;
    int *Shape;
    int shape[4] = {40,1,128,0};

    // Alloc memory for CPU
    input = (float*)malloc(1024*1024*sizeof(float));
    weight = (float*)malloc(1024*1024*sizeof(float));
    output = (float*)malloc(1024*1024*sizeof(float));

    // Array initialization
    /*for(int i=0; i<16; i++) {
        input[i] = (float)rand();
        weight[i] = (float)rand();
    }*/

    double start, exetime;
    start = seconds();

    // Alloc memory for GPU
    hipMalloc((void**)&Input, 1024*1024*sizeof(float));
    hipMalloc((void**)&Weight, 1024*1024*sizeof(float));
    hipMalloc((void**)&Output, 1024*1024*sizeof(float));
    hipMalloc((void**)&Shape, 4*sizeof(int));

    // Move data from CPU to GPU
    hipMemcpy(Input, input, shape[0]*shape[1]*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Weight, weight, shape[1]*shape[2]*sizeof(float), hipMemcpyHostToDevice);
    //cudaMemcpy(Output, output, shape[0]*shape[2]*sizeof(float), cudaMemcpyHostToDevice);
    hipMemcpy(Shape, shape, 4*sizeof(int), hipMemcpyHostToDevice);

    // Configuration of kernels, basic block has 64 threads
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(shape[0]/8+1,shape[2]/8+1,1);

    // Kernel execution
    CUDA_matrix_multiplication<<<dimGrid, dimBlock>>>(Input, Weight, Output, Shape);

    // Move data from GPU to CPU
    hipMemcpy(output, Output, shape[0]*shape[2]*sizeof(float), hipMemcpyDeviceToHost);
    
    shape[0] = 128; shape[1] = 1; shape[2] =1; shape[3] = 0;
    hipMemcpy(Input, output, shape[0]*shape[1]*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Shape, shape, 4*sizeof(int), hipMemcpyHostToDevice);

    CUDA_matrix_multiplication<<<dimGrid, dimBlock>>>(Input, Weight, Output, Shape);
    hipMemcpy(output, Output, shape[0]*shape[2]*sizeof(float), hipMemcpyDeviceToHost);

    exetime = seconds() - start;
    printf("Time used:%f us\n", exetime*1000000);


    for(int i=0;i<3; i++) {
        for(int j=0; j<3; j++) {
            cout<<output[i*3+j]<<" ";
        }
        cout<<endl;
    }

    // Free memory
    free(input);
    free(weight);
    free(output);
    hipFree(Input);
    hipFree(Weight);
    hipFree(Output);
    hipFree(Shape);
    
    return 0;
}
#endif