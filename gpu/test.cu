
#include <iostream>
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>

//#define DEVICE_TEST

using namespace std;

#ifdef DEVICE_TEST
int main()
{
    int deviceCount;
    cudaGetDeviceCount(&deviceCount);
    for(int i=0;i<deviceCount;i++)
    {
        cudaDeviceProp devProp;
        cudaGetDeviceProperties(&devProp, i);
        std::cout << "使用GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << "设备全局内存总量： " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
        std::cout << "SM的数量：" << devProp.multiProcessorCount << std::endl;
        std::cout << "每个线程块的共享内存大小：" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
        std::cout << "每个线程块的最大线程数：" << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "设备上一个线程块（Block）种可用的32位寄存器数量： " << devProp.regsPerBlock << std::endl;
        std::cout << "每个EM的最大线程数：" << devProp.maxThreadsPerMultiProcessor << std::endl;
        std::cout << "每个EM的最大线程束数：" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
        std::cout << "设备上多处理器的数量： " << devProp.multiProcessorCount << std::endl;
        std::cout << "======================================================" << std::endl;     
        
    }
    return 0;
}
#else

__global__ void CUDA_matrix_multiplication1(float Input[], float Weight[], float Output[]) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int colum = blockIdx.x*blockDim.x + threadIdx.x;
    
    float sum = 0;
    if(row<128 && colum<1) {
        for (int i=0; i<40; i++) {
            sum += Weight[row*40 + i] * Input[i*1 + colum];
        }
        Output[row*1 + colum] = sum;
    }
}

__global__ void CUDA_matrix_multiplication2(float Input[], float Weight[], float Output[]) {
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int colum = blockIdx.x*blockDim.x + threadIdx.x;
    
    float sum = 0;
    if(row<1 && colum<1) {
        for (int i=0; i<128; i++) {
            sum += Input[row*128 + i] * Weight[i*1 + colum];
        }
        Output[row*1 + colum] = sum;
    }
}

int main() {

    float *input, *weight, *output;
    float *Input, *Weight, *Output;

    // Alloc memory for CPU
    //cudaHostAlloc((void **) &input, sizeof(float)*1024*1024, cudaHostAllocDefault);
    //cudaHostAlloc((void **) &weight, sizeof(float)*1024*1024, cudaHostAllocDefault);
    //cudaHostAlloc((void **) &output, sizeof(float)*1024*1024, cudaHostAllocDefault);
    input = (float*)malloc(1024*1024*sizeof(float));
    weight = (float*)malloc(1024*1024*sizeof(float));
    output = (float*)malloc(1024*1024*sizeof(float));

    // Array initialization
    /*for(int i=0; i<3; i++) {
        for(int j=0; j<3; j++) {
            input[i*3+j] = i;
            weight[i*3+j] = 1;
        }
    }*/


    // Alloc memory for GPU
    hipMalloc((void**)&Input, 1024*1024*sizeof(float));
    hipMalloc((void**)&Weight, 1024*1024*sizeof(float));
    hipMalloc((void**)&Output, 1024*1024*sizeof(float));

    // Move data from CPU to GPU
    hipMemcpy(Input, input, 40*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Weight, weight, 128*40*sizeof(float), hipMemcpyHostToDevice);
    //cudaMemcpy(Output, output, shape[0]*shape[2]*sizeof(float), cudaMemcpyHostToDevice);

    // Configuration of kernels, basic block has 64 threads
    dim3 dimBlock(16,16,1);
    dim3 dimGrid(1/16+1,1/16+1,1);

    // Kernel execution
    CUDA_matrix_multiplication1<<<dimGrid, dimBlock>>>(Input, Weight, Output);
    CUDA_matrix_multiplication2<<<dimGrid, dimBlock>>>(Output, Weight, Input);

    // Move data from GPU to CPU
    //cudaMemcpy(output, Output, shape[0]*shape[2]*sizeof(float), cudaMemcpyDeviceToHost);
    
    //cudaMemcpy(Input, output, shape[0]*shape[1]*sizeof(float), cudaMemcpyHostToDevice);
    //cudaMemcpy(Shape, shape, 4*sizeof(int), cudaMemcpyHostToDevice);

    //CUDA_matrix_multiplication<<<dimGrid, dimBlock>>>(Input, Weight, Output, Shape);

    hipMemcpy(input, Input, 1*sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    //cudaFreeHost(input);
    //cudaFreeHost(weight);
    //cudaFreeHost(output);
    free(input);
    free(weight);
    free(output);
    hipFree(Input);
    hipFree(Weight);
    hipFree(Output);
    
    return 0;
}
#endif